#include "hip/hip_runtime.h"
#include "tools.cuh" // 引入array.cuh头文件

#include <random>
#include <vector>
#include <unordered_set>
#include <cmath>
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/unique.h>
#include <thrust/sort.h>
#include <chrono> 
// #include <pybind11/pybind11.h>
// #include <pybind11/numpy.h>

// namespace py = pybind11;


// // GPU kernel to add particles to corresponding cells
// __global__ void add_particles_to_cells_gpu(const int* particle_inds, Dynamic2DArray<int>& cell_list, int num_particles) {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
//     if (tid < num_particles) {
//         int particle_ind = particle_inds[tid];
//         cell_list.push_back(particle_ind, tid);
//         // atomicAdd(&cell_list[particle_ind], tid);
//     }
// }

// // GPU kernel to build neighbor relationships considering rc
// __global__ void build_neighbor_relationships_gpu(const double* inputs, const int* particle_inds,
//                                                  const int* cell_list, int* particle_list,
//                                                  int num_particles, double cube_size_x, double cube_size_y, double cube_size_z,
//                                                  double grid_size_x, double grid_size_y, double grid_size_z,
//                                                  double rc) {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
//     if (tid < num_particles) {
//         double particle_xyz[3];
//         particle_xyz[0] = inputs[tid];
//         particle_xyz[1] = inputs[tid + num_particles];
//         particle_xyz[2] = inputs[tid + 2 * num_particles];

//         int particle_ind = particle_inds[tid];
//         // 继续原有逻辑
//     }
// }

class GB_NBL_GPU_cube {
public:
    GB_NBL_GPU_cube(const thrust::host_vector<double>& c_size, int num_particles, double cut_off_radius)
        : rc(cut_off_radius), num_particles(num_particles)
    {
        this->cube_size = c_size;

        lc = thrust::device_vector<int>(3);
        lc[0] = static_cast<int>(cube_size[0] / cut_off_radius);
        lc[1] = static_cast<int>(cube_size[1] / cut_off_radius);
        lc[2] = static_cast<int>(cube_size[2] / cut_off_radius);

        grid_size = thrust::device_vector<double>(3);
        grid_size[0] = cube_size[0] / lc[0];
        grid_size[1] = cube_size[1] / lc[1];
        grid_size[2] = cube_size[2] / lc[2];

        num_cells = lc[0] * lc[1] * lc[2];

        particle_list.resize(num_particles, 100);
        cell_list.resize(num_cells, 100);
    }

    // Convert xyz to cell indices using CUDA (GPU)
    thrust::device_vector<int> xyz2ind_gpu(Dynamic2DArray<double>& xyz) {
        thrust::device_vector<int> cell_indices(xyz.max_rows);
        for (int i = 0; i < xyz.max_rows; ++i) {
            thrust::device_vector<int> cell_index(3);

            for (int j = 0; j < 3; j++) {
                double x = fmod(xyz.getElement(i, j), cube_size[j]); // Access data on GPU

                if (x < 0) x += cube_size[j];

                cell_index[j] = static_cast<int>(floor(x / grid_size[j]));
            }
            cell_indices[i] = cell_index[0] * lc[1] * lc[2] + cell_index[1] * lc[2] + cell_index[2];
        }
        return cell_indices;
    }


    // Convert cell indices to xyz using CUDA (GPU)
    Dynamic2DArray<double> ind2xyz_gpu(const thrust::device_vector<int>& ind) {
        Dynamic2DArray<double> xyz(ind.size(), 3);

        for (int i = 0; i < ind.size(); ++i) {
            thrust::device_vector<double> point(3);
            int index = ind[i];

            point[0] = (index / (lc[1] * lc[2])) * grid_size[0] + grid_size[0] / 2;
            point[1] = (index / lc[2]) * grid_size[1] + grid_size[1] / 2;
            point[2] = (index % lc[2]) * grid_size[2] + grid_size[2] / 2;

            xyz.push_back(i, fmod(point[0], cube_size[0]));
            xyz.push_back(i, fmod(point[1], cube_size[1]));
            xyz.push_back(i, fmod(point[2], cube_size[2]));

            
            // std::cout << "___________" << std::endl;
            // std::cout << "0: " << xyz.helper[0] << " 1: " << xyz.helper[1] << " 2: " << xyz.helper[2] << std::endl;
            // xyz.print();
            // break;
        }
        return xyz;
    }

    // Get the minimum difference between two points using CUDA (GPU)
    thrust::device_vector<double> get_min_diff_gpu(const thrust::device_vector<double>& xyz1, const thrust::device_vector<double>& xyz2) {
        thrust::device_vector<double> difference(3);
        for (int i = 0; i < 3; i++) {
            double diff = xyz2[i] - xyz1[i];
            diff = std::fmod(diff + cube_size[i] / 2, cube_size[i]);
            if (diff < 0) diff += cube_size[i];
            diff -= cube_size[i] / 2;
            difference[i] = diff;
        }
        return difference;
    }

    // Get neighbor cells using CUDA (GPU)
    thrust::device_vector<int> get_neighbor_cells_gpu(int cell_ind) {
        thrust::device_vector<int> cell_ind_list(1);
        cell_ind_list[0] = cell_ind;
        Dynamic2DArray<double> xyz = ind2xyz_gpu(cell_ind_list);

        thrust::device_vector<int> adjacent_cells;
        for (int di = -1; di <= 1; di++) {
            for (int dj = -1; dj <= 1; dj++) {
                for (int dk = -1; dk <= 1; dk++) {
                    double ni = static_cast<int>(xyz.getElement(0, 0)) + di * static_cast<int>(grid_size[0]);
                    double nj = static_cast<int>(xyz.getElement(0, 1)) + dj * static_cast<int>(grid_size[1]);
                    double nk = static_cast<int>(xyz.getElement(0, 2)) + dk * static_cast<int>(grid_size[2]);
                    Dynamic2DArray<double> point(1, 3);
                    point.push_back(0, ni);
                    point.push_back(0, nj);
                    point.push_back(0, nk);
                    thrust::device_vector<int> ind_temp = xyz2ind_gpu(point);

                    adjacent_cells.push_back(ind_temp[0]);
                }
            }
        }

        thrust::device_vector<int> result = vector2set_int(adjacent_cells);

        return result;
    }


    thrust::device_vector<int> get_neighbors(int particle_seq) {
        int neighbor_njm = particle_list.helper[particle_seq];
        thrust::device_vector<int> res(neighbor_njm);

        res = particle_list.getVector(particle_seq);   

        return res;
    }

    // Convert constructor to gpu with two different GPU kernel
    void constructor_gpu(Dynamic2DArray<double>& inputs) {
        // Step 1: Convert xyz to cell indices using CUDA (GPU)
        thrust::device_vector<int> particle_inds = xyz2ind_gpu(inputs);

        // Step 2: Add particles to corresponding cells
        for (int i = 0; i < num_particles; i++) {
            cell_list.push_back(particle_inds[i], i);
            // std::cout << "temp" << particle_inds[i] << " " << i << std::endl;
        }

        // // Step 2: Add particles to corresponding cells
        // thrust::device_vector<int> device_cell_list(num_cells, -1);
        // add_particles_to_cells_gpu<<<num_blocks, block_size>>>(particle_inds.data().get(),
        //                                                        cell_list,
        //                                                        num_particles);

        // Step 3: Build neighbor relationships considering rc
        for (int particle_seq = 0; particle_seq < num_particles; particle_seq++) {
            thrust::device_vector<double> particle_xyz = inputs.getVector(particle_seq);

            int particle_ind = particle_inds[particle_seq];
            thrust::device_vector<int> adjacent_cells = get_neighbor_cells_gpu(particle_ind);

            for (int i = 0; i < adjacent_cells.size(); i++) {
                thrust::device_vector<int> neighbor_particles_temp = cell_list.getVector(adjacent_cells[i]);
                if (neighbor_particles_temp.size()) {
                    for (int j = 0; j < neighbor_particles_temp.size(); j++) {
                        int neighbor_particle_seq = neighbor_particles_temp[j];
                        if (neighbor_particle_seq == particle_seq) continue;
                        thrust::device_vector<double> neighbor_xyz = inputs.getVector(neighbor_particle_seq);
                        thrust::device_vector<double> diff = get_min_diff_gpu(particle_xyz, neighbor_xyz);

                        double distance = 0;
                        for (int k = 0; k < 3; k++) {
                            distance += diff[i] * diff[i];
                        }

                        if (distance - rc * rc < 1e-10) {
                            particle_list.push_back(particle_seq, neighbor_particle_seq);
                        }
                    }
                }
            }
        }
        // // Step 3: Build neighbor relationships considering rc
        // thrust::device_vector<int> device_particle_list(num_particles, -1);
        // build_neighbor_relationships_gpu<<<num_blocks, block_size>>>(inputs.data().get(),
        //                                                              particle_inds.data().get(),
        //                                                              device_cell_list.data().get(),
        //                                                              device_particle_list.data().get(),
        //                                                              num_particles,
        //                                                              cube_size[0], cube_size[1], cube_size[2],
        //                                                              grid_size[0], grid_size[1], grid_size[2],
        //                                                              rc);

        // // Copy the result back to the host
        // thrust::copy(device_cell_list.begin(), device_cell_list.end(), cell_list.begin());
        // thrust::copy(device_particle_list.begin(), device_particle_list.end(), particle_list.begin());
    }

    double rc;                                  // the cut-off radius
    int num_particles;                          // the number of particles
    int num_cells;                              // the number of cells

    thrust::device_vector<double> cube_size;     // cube domain
    thrust::device_vector<int> lc;              // cell size
    thrust::device_vector<double> grid_size;    // grid size

    Dynamic2DArray<int> particle_list; // Dynamic2DArray represents particle_list and cell_list
    Dynamic2DArray<int> cell_list; 

    // const int num_blocks = 12;
    // const int block_size = 1024;
};


double generate_random_double(double min_val, double max_val) {
    static std::random_device rd;
    static std::mt19937 gen(rd());
    std::uniform_real_distribution<double> dis(min_val, max_val);
    return dis(gen);
}


// 绑定 Dynamic2DArray 类模板到 Python
template <typename T>
void bind_Dynamic2DArray(py::module& m, const std::string& name) {
    // 定义中间函数来明确模板类型
    auto class_fn = [](py::module& m, const std::string& name) {
        return py::class_<Dynamic2DArray<T>>(m, name)
            .def(py::init<>())
            .def(py::init<int, int>())
            .def("push_back", &Dynamic2DArray<T>::push_back)
            .def("getElement", &Dynamic2DArray<T>::getElement)
            .def("getVector", &Dynamic2DArray<T>::getVector)
            .def("resize", &Dynamic2DArray<T>::resize)
            .def("print", (void (Dynamic2DArray<T>::*)()) &Dynamic2DArray<T>::print)
            .def("print", (void (Dynamic2DArray<T>::*)(int)) &Dynamic2DArray<T>::print);
    };

    // 调用中间函数
    class_fn(m, name);
}

PYBIND11_MODULE(grid_based_nbl_cpp, m) {
    // // Dynamic2DArray类模板的Pybind11绑定
    // bind_Dynamic2DArray<double>(m, "Dynamic2DArrayDouble");
    // bind_Dynamic2DArray<int>(m, "Dynamic2DArrayInt");

    // // 辅助函数vector2set_int的Pybind11绑定
    // m.def("vector2set_int", &vector2set_int, "Convert a vector to a set (remove duplicates) using Thrust.");

    // GB_NBL_GPU_cube类的Pybind11绑定
    py::class_<GB_NBL_GPU_cube>(m, "GB_NBL_GPU_cube")
        .def(py::init<const thrust::host_vector<double>&, int, double>())
        .def("constructor_gpu_wrapper", &GB_NBL_GPU_cube::constructor_gpu)
        .def("get_neighbors", &GB_NBL_GPU_cube::get_neighbors, py::arg("particle_seq"));
}




int main() {
    // test xyz - index转换
    thrust::host_vector<double> c_size(3);
    for (int i = 0; i < 3; i++) {
        c_size[i] = 10.0; 
    }

    int num_particles = 3;
    double cut_off_radius = 1.0;

    // call GB_NBL_GPU_cube constructor with thrust::device_vector
    GB_NBL_GPU_cube lc_cube(c_size, num_particles, cut_off_radius);

    // convert xyz to thrust::device_vector of Dynamic2DArray
    std::vector<std::vector<double>> xyz_temp = { {1.5, 2.2, 3.7}, {4.1, 5.9, 6.3}, {7.8, 8.4, 9.2} };
    Dynamic2DArray<double> xyz(3, 3);
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            xyz.push_back(i, xyz_temp[i][j]);
        }
    }

    // call the GPU wrapper functions to perform computations on the GPU
    thrust::device_vector<int> cell_indices = lc_cube.xyz2ind_gpu(xyz);
    thrust::host_vector<int> host_cell_indices = cell_indices;
    std::cout << "Cell indices:" << std::endl;
    for (const auto& index : host_cell_indices) {
        std::cout << index << " ";
    }
    std::cout << std::endl;


    // call the GPU wrapper function for ind2xyz_gpu
    std::cout << "Restore: " << std::endl;
    Dynamic2DArray<double> device_restored_xyz = lc_cube.ind2xyz_gpu(cell_indices);
    device_restored_xyz.print();
    thrust::device_vector<double> temp = device_restored_xyz.getVector(0);
    for (int i = 0; i < temp.size(); i++) {
        std::cout << temp[i] << " ";
    }
    std::cout << "sighn" << std::endl;

    // check the distance calculation
    std::vector<double> cube_size2 = { 20, 20, 20 };
    thrust::device_vector<double> cube_size2_dev(3);
    for(int i = 0; i < 3; i++) {
        cube_size2_dev[i] = cube_size2[i];
    }

    int num_particles2 = 800;
    double cut_off_radius2 = 2.0;
    GB_NBL_GPU_cube lc_cube2(cube_size2, num_particles2, cut_off_radius2);

    std::vector<double> p1 = {19.608,  7.38 ,  6.096}; 
    thrust::device_vector<double> p1_dev(3);
    for(int i = 0; i < 3; i++) {
        p1_dev[i] = p1[i];
    }

    std::vector<double> p2 = {0.   , 9.012, 6.944};
    thrust::device_vector<double> p2_dev(3);
    for(int i = 0; i < 3; i++) {
        p2_dev[i] = p2[i];
    }

    thrust::device_vector<double> res = lc_cube2.get_min_diff_gpu(p1, p2);
    std::cout << "Real value: [" << 0.392 << " " << 1.632 << " " << 0.848 << "]" << std::endl;
    std::cout << "Calculated value: [" << res[0] << " " << res[1] << "" << res[2] << "]" << std::endl;
    


    // check get_neighbor_cells_gpu
    thrust::host_vector<double> c_size(3);
    for (int i = 0; i < 3; i++) {
        c_size[i] = 80.0; 
    }

    int num_particles = 5000;
    double cut_off_radius = 2.0;

    // call GB_NBL_GPU_cube constructor with thrust::device_vector
    GB_NBL_GPU_cube lc_cube3(c_size, num_particles, cut_off_radius);

    // thrust::device_vector res = lc_cube3.get_neighbor_cells_gpu(1);
    // for(int i = 0; i < res.size(); i++) {
    //     std::cout << res[i] << " ";
    // }

    // check the constructor
    std::vector<std::vector<double>> xyz_temp;
    for (int i = 0; i < num_particles; i++) {
        double x = generate_random_double(0.0, 80.0);
        double y = generate_random_double(0.0, 80.0);
        double z = generate_random_double(0.0, 80.0);
        xyz_temp.push_back({x, y, z});
    }

    // put xyz_temp into Dynamic2DArray<double> xyz 
    Dynamic2DArray<double> xyz(num_particles, 3);
    for (int i = 0; i < num_particles; i++) {
        for (int j = 0; j < 3; j++) {
            xyz.push_back(i, xyz_temp[i][j]);
        }
    }

    std::cout << "Data Preparation Ready!" << std::endl;

    // start time
    auto start_time = std::chrono::high_resolution_clock::now();

    lc_cube3.constructor_gpu(xyz);
    // end time
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end_time - start_time;

    std::cout << "Time: " << duration.count() << " s" << std::endl;




    // test get_neighbors
    thrust::device_vector res_neighbor = lc_cube3.get_neighbors(0);
    for (int i = 0; i < res_neighbor.size(); i++){
        std::cout << res_neighbor[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Thrust version: " << THRUST_MAJOR_VERSION << "." << THRUST_MINOR_VERSION << std::endl;


    return 0;
}
