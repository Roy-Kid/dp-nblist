#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <sstream>
#include <chrono>
#include "cuda_nblist.h"
#include "box.h"
#include "vec3.cpp"

// CUDA相关头文件
#include <hip/hip_runtime.h>
#include <>

namespace dpnblist
{
    /*
    * to get the neighbor cell index of each cell, and store them in nebcell_list
    * param: offset_vec: the offset vector of each neighbor cell
    * param: ncells: the number of cells
    * param: n_nebcells: the number of neighbor cells
    * param: L: the box length of each dimension
    */
    __global__ void get_neb(int* offset_vec, size_t *ncells, size_t *n_nebcells, int *L, int* nebcell) {
        int tid = blockIdx.x * blockDim.x + threadIdx.x;

        if (tid < *ncells) {
            // int L[3] = {4, 4, 4};
            int cell_veci[3];
            int offset_veci[3];
            float neb_vec[3];
            float wrap_neb_vec[3];
            int round_wrap_neb_vec[3];
            float f[3];
            float wrapped_f[3];

            cell_veci[0] = tid / (L[1] * L[2]);
            cell_veci[1] = (tid - cell_veci[0] * L[1] * L[2]) / L[2];
            cell_veci[2] = tid - cell_veci[0] * L[1] * L[2] - cell_veci[1] * L[2];

            for (int i = 0; i < *n_nebcells; ++i) {
                for (int j = 0; j < 3; ++j) {
                    offset_veci[j] = offset_vec[i * 3 + j];
                    neb_vec[j] = cell_veci[j] + offset_veci[j];
                }
                for (int k = 0; k < 3; ++k) {
                    f[k] = neb_vec[k] / L[k];
                    wrapped_f[k] = f[k] - std::floor(f[k] + 0.000001);
                    wrap_neb_vec[k] = wrapped_f[k] * L[k];
                    round_wrap_neb_vec[k] = round(wrap_neb_vec[k]);
                }
                
                nebcell[*n_nebcells * tid + i] = round_wrap_neb_vec[0] * L[1] * L[2] + round_wrap_neb_vec[1] * L[2] + round_wrap_neb_vec[2];
            }
        }
    }
    /*
    * to build the linked list, and store the head, lscl, atom_cellindex, cell_count
    * param: xyz_dev: the coordinates of all atoms
    * param: head_dev: the head atom of each cell
    * param: lscl_dev: the next atom of each cell
    * param: atom_cellindex: the cell index of each atom
    * param: cell_count_dev: the number of atoms in each cell
    * param: r_cutoff: the cutoff radius
    * param: L: the box length of each dimension
    * param: natoms: the number of atoms
    */
    __global__ void build_linked_list_kernel(float *xyz_dev, int *head_dev, int *lscl_dev, int *atom_cellindex, int *cell_count_dev, float *_r_cutoff, int *L, size_t *natoms) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < *natoms) {
            int cell_index_vec[3];
            for (int j = 0; j < 3; j++) {
                cell_index_vec[j] = xyz_dev[i * 3 + j] / *_r_cutoff;
                if (cell_index_vec[j] == L[j])
                    cell_index_vec[j] = cell_index_vec[j] - 1;
            }
            int cell_index = cell_index_vec[0] * L[1] * L[2] + cell_index_vec[1] * L[2] + cell_index_vec[2];
            atom_cellindex[i] = cell_index;
            lscl_dev[i] = atomicExch(&head_dev[cell_index], i);
            atomicAdd(&cell_count_dev[cell_index], 1);
        }
    }

    /*
    * to calculate the distance between two atoms, and return the square of the distance
    * param: pos_i: the coordinates of atom i
    * param: pos_j: the coordinates of atom j
    * param: length: the box length of each dimension
    */
    __device__ float calc_distance(float *pos_i, float *pos_j, float *length) {
        float difference[3] = {0.0, 0.0, 0.0};
        float diff = 0.0;

        for (int i = 0; i < 3; i++) {
            float dri = pos_i[i] - pos_j[i];
            diff = fmodf((dri + length[i] / 2), length[i]);
            if (diff < 0) diff += length[i];
            diff -= length[i] / 2;
            difference[i] = diff;
        }

        return (difference[0]*difference[0] + difference[1]*difference[1] + difference[2]*difference[2]);
    }

    /*
    * to build the neighbor list array, and store the neighbor list of each atom to neighborListArray
    * param: natoms: the number of atoms
    * param: xyz: the coordinates of all atoms
    * param: head: the head atom of each cell
    * param: lscl: the next atom of each cell
    * param: atom_cellindex: the cell index of each atom
    * param: cell_count: the number of atoms in each cell
    * param: nebcell: the neighbor cell index of each cell
    * param: L: the box length of each dimension
    * param: box_length: the cell number of each dimension
    * param: r_cutoff2: the square of cutoff radius
    * param: neighborListArray: the neighbor list of each atom
    */
    __global__ void  buildListArray(size_t *natoms, float *xyz, int *head, int *lscl, int *atom_cellindex, int *cell_atoms_count, int *nebcell, int *L, float *box_length, float *r_cutoff2, int *neighborListArray){
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if (tid < *natoms){
            size_t count = 0;                                                   //记录这是第几个邻居原子
            float pos_i[3] = {xyz[tid*3], xyz[tid*3+1], xyz[tid*3+2]};          //原子坐标
            size_t cell_index = atom_cellindex[tid];                            //原子属于哪个单元
            size_t neighbor_cell = 0;                                           //单元的邻居单元索引
            for (int i = cell_index * 27; i < cell_index * 27 + 27; ++i)        // Loop through the neighboring cells
            {                                                                   
                neighbor_cell = nebcell[i];                                     //27个邻居单元索引
                size_t num_atoms = cell_atoms_count[neighbor_cell];             //单元有几个原子
                size_t atom_id = head[neighbor_cell];                           //单元的第一个原子
                for (size_t j = 0; j < num_atoms; ++j)                          //循环单元内所有的原子
                {
                    float pos_j[3] = {xyz[atom_id*3], xyz[atom_id*3+1], xyz[atom_id*3+2]};
                    float r = calc_distance(pos_i, pos_j, box_length);
                    if (r < *r_cutoff2)
                    {
                        neighborListArray[tid * 100 + count] = atom_id;                   // Add j to the neighbor list of i
                        count++;
                    }
                    atom_id = lscl[atom_id];
                }
            }
        }
    }

    /*
    * the constructor of class CudaCellList, to initialize the parameters
    * param: box: the box of the system
    * param: r_cutoff: the cutoff radius
    * param: skin: the skin of the system
    * initialize the parameters of the system and allocate the memory of the device
    * param: r_cutoff: the cutoff radius
    * param: skin: the skin of the system
    * param: d_box_len: the box length of each dimension
    * param: d_cell_len: the cell number of each dimension
    * param: d_ncells: the number of cells
    */
    
    CudaCellList::CudaCellList(Box *box, float r_cutoff, float skin):_box(box), _r_cutoff(r_cutoff+skin),_skin(skin)
	{
	  Vec3<float> box_length = box->get_lengths();
        for (int i = 0; i < 3; ++i) {
            _box_len[i] = box_length[i];
        }
        //_r_cutoff = r_cutoff + skin;
        hipMalloc((void**)&d_r_cutoff, sizeof(float));
        hipMemcpy(d_r_cutoff, &_r_cutoff, sizeof(float), hipMemcpyHostToDevice);

        //_skin = skin;
        hipMalloc((void**)&d_skin, sizeof(float));
        hipMemcpy(d_skin, &_skin, sizeof(float), hipMemcpyHostToDevice);

        for (int i = 0; i < 3; ++i) {
            _cell_len[i] = _box_len[i]/_r_cutoff;
        }
        _ncells = _cell_len[0] * _cell_len[1] * _cell_len[2];

        hipMalloc((void**)&d_box_len, 3 * sizeof(float));
        hipMemcpy(d_box_len, _box_len, 3 * sizeof(float), hipMemcpyHostToDevice);

        hipMalloc((void**)&d_cell_len, 3 * sizeof(int));
        hipMemcpy(d_cell_len, _cell_len, 3 * sizeof(int), hipMemcpyHostToDevice);

        hipMalloc((void**)&d_ncells, sizeof(size_t));
        hipMemcpy(d_ncells, &_ncells, sizeof(size_t), hipMemcpyHostToDevice);

    }

    /*
    * to build the neighbor list array, and initialize the parameters and allocate the memory of the device
    * variable: d_natoms: the number of atoms
    * variable: d_off_set_vec: the offset vector of each neighbor cell
    * variable: d_n_nebcells: the number of neighbor cells
    * variable: d_nebcell_list: the neighbor cell index of each cell
    * run kernel get_neb to get the neighbor cell index of each cell
    * run update(xyz) to build the neighbor list array
    */
    void CudaCellList::build(std::vector<std::vector<float>> &xyz) {
        auto start_time = std::chrono::high_resolution_clock::now();

        _natoms = xyz.size();
        hipMalloc((void**)&d_natoms, sizeof(size_t));
        hipMemcpy(d_natoms, &_natoms, sizeof(size_t), hipMemcpyHostToDevice);

        
        std::vector<std::vector<int>> off_set_vec = {
            {-1, -1, -1},{-1, -1, 0},{-1, -1, 1},{-1, 0, -1},{-1, 0, 0},{-1, 0, 1},{-1, 1, -1},{-1, 1, 0},{-1, 1, 1},
            {0, -1, -1},{0, -1, 0},{0, -1, 1},{0, 0, -1},{0, 0, 0},{0, 0, 1},{0, 1, -1},{0, 1, 0},{0, 1, 1},
            {1, -1, -1},{1, -1, 0},{1, -1, 1},{1, 0, -1},{1, 0, 0},{1, 0, 1},{1, 1, -1},{1, 1, 0},{1, 1, 1}
        };
        size_t n_nebcells = off_set_vec.size();
        size_t *d_n_nebcells;
        hipMalloc((void**)&d_n_nebcells, sizeof(size_t));
        hipMemcpy(d_n_nebcells, &n_nebcells, sizeof(size_t), hipMemcpyHostToDevice);

        int off_set_vec_1d[n_nebcells * 3];
        for (int i = 0; i < n_nebcells; ++i) {
            for (int j = 0; j< 3; ++j) {
                off_set_vec_1d[i * 3 + j] = off_set_vec[i][j];
            }
        }
        int *d_off_set_vec_1d;
        hipMalloc((void**)&d_off_set_vec_1d, n_nebcells * 3 * sizeof(int));
        hipMemcpy(d_off_set_vec_1d, off_set_vec_1d, n_nebcells * 3 * sizeof(int), hipMemcpyHostToDevice);
        
        hipMalloc((void**)&d_nebcell_list, _ncells * n_nebcells * sizeof(int));
        int threadsPerBlock = 256;
        int blocksPerGrid = (_ncells + threadsPerBlock - 1) / threadsPerBlock;
        //////////
        auto neb_start = std::chrono::high_resolution_clock::now();
        get_neb<<<blocksPerGrid, threadsPerBlock>>>(d_off_set_vec_1d, d_ncells, d_n_nebcells, d_cell_len, d_nebcell_list);
        auto neb_end = std::chrono::high_resolution_clock::now();
        auto neb_duration = std::chrono::duration_cast<std::chrono::microseconds>(neb_end - neb_start);
        std::cout << "Time taken by get nebcell: " << neb_duration.count() << " microseconds" << std::endl;
        //////////
        hipFree(d_off_set_vec_1d);

        // int *d_head, *d_lscl, *d_atom_cellindex, *d_cell_atoms_count;
        hipMalloc((void**)&d_head, _ncells * sizeof(int));                      // head[i] is the first atom in cell i
        hipMalloc((void**)&d_lscl, _natoms * sizeof(int));                      // lscl is the atom linked list, lscl[i] is the next atom in the cell
        hipMalloc((void**)&d_atom_cellindex, _natoms * sizeof(int));            // atom in which cell
        hipMalloc((void**)&d_cell_atoms_count, _ncells * sizeof(int));          // cell_atoms_count[i] is the number of atoms in cell i
        
        _neighborListArray = new int[_natoms * 100];
        update(xyz);

        auto end_time = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
        time_cost = duration.count();
        std::cout << "Time taken by cuda: " << time_cost << " milliseconds" << std::endl;
    }

    /*
    * to update the neighbor list array, and allocate the memory of the device
    * variable: d_xyz_1d: the coordinates of all atoms
    * variable: d_head: the head atom of each cell
    * variable: d_lscl: the next atom of each cell
    * variable: d_atom_cellindex: the cell index of each atom
    * variable: d_cell_atoms_count: the number of atoms in each cell
    * run kernel build_linked_list_kernel to build the linked list
    * run kernel buildListArray to build the neighbor list array
    */
    void CudaCellList::update(std::vector<std::vector<float>> &xyz) {

        std::vector<Vec3<float>> wrap_xyz;
        for (int i = 0; i < _natoms; ++i) {
            wrap_xyz.emplace_back(xyz[i][0], xyz[i][1], xyz[i][2]);
        }
	    wrap_xyz = _box->wrap(wrap_xyz);
        float *xyz_1d = new float[_natoms * 3];
        // float xyz_1d[_natoms*3];
        for (int i = 0; i < _natoms; ++i) {
            for (int j = 0; j< 3; ++j) {
                xyz_1d[i * 3 + j] = wrap_xyz[i][j];
            }
        }

        float *d_xyz_1d;
        hipMalloc((void**)&d_xyz_1d, _natoms * 3 * sizeof(float));
        hipMemcpy(d_xyz_1d, xyz_1d, _natoms * 3 * sizeof(float), hipMemcpyHostToDevice);

        // int *d_head, *d_lscl, *d_atom_cellindex, *d_cell_atoms_count;
        _cell_atoms_count = new int[_ncells];
        for (int i = 0; i < _ncells; ++i) {
            _cell_atoms_count[i] = 0;
        }
        hipMemcpy(d_cell_atoms_count, _cell_atoms_count, _ncells * sizeof(int), hipMemcpyHostToDevice);
        // auto start_time = std::chrono::high_resolution_clock::now();
        int threadsPerBlock = 256;
        int blocksPerGrid = (_natoms + threadsPerBlock - 1) / threadsPerBlock;

        //////////
        auto linked_list_start = std::chrono::high_resolution_clock::now();
        build_linked_list_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_xyz_1d, d_head, d_lscl, d_atom_cellindex, d_cell_atoms_count, d_r_cutoff, d_cell_len, d_natoms);
        auto linked_list_end = std::chrono::high_resolution_clock::now();
        auto linked_list_duration = std::chrono::duration_cast<std::chrono::microseconds>(linked_list_end - linked_list_start);
        std::cout << "Time taken by build_linked_list: " << linked_list_duration.count() << " microseconds" << std::endl;
        //////////

        float r_cutoff2 = _r_cutoff * _r_cutoff;
        float *d_r_cutoff2;
        hipMalloc((void**)&d_r_cutoff2, sizeof(float));
        hipMemcpy(d_r_cutoff2, &r_cutoff2, sizeof(float), hipMemcpyHostToDevice);

        int *temp_neighborListArray = new int[_natoms * 100]; // neighborListArray is the neighbor list of atoms, has a maximum 100 neighbors for each atom
        for (int i = 0; i < _natoms * 100; ++i) {
            temp_neighborListArray[i] = -1;
        }

        int *d_neighborListArray;
        hipMalloc((void**)&d_neighborListArray, _natoms * 100 * sizeof(int));
        hipMemcpy(d_neighborListArray, temp_neighborListArray, _natoms * 100 * sizeof(int), hipMemcpyHostToDevice);

        blocksPerGrid = (_natoms + threadsPerBlock - 1) / threadsPerBlock;

        //////////
        auto build_start = std::chrono::high_resolution_clock::now();
        buildListArray<<<blocksPerGrid, threadsPerBlock>>>(d_natoms, d_xyz_1d, d_head, d_lscl, d_atom_cellindex, d_cell_atoms_count, d_nebcell_list, d_cell_len, d_box_len, d_r_cutoff2, d_neighborListArray);
        auto build_end = std::chrono::high_resolution_clock::now();
        auto build_duration = std::chrono::duration_cast<std::chrono::microseconds>(build_end - build_start);
        std::cout << "Time taken by build list array: " << build_duration.count() << " microseconds" << std::endl;
        //////////

        //////////
        auto copy_start = std::chrono::high_resolution_clock::now();
        hipMemcpy(_neighborListArray, d_neighborListArray, _natoms * 100 * sizeof(int), hipMemcpyDeviceToHost);
        auto copy_end = std::chrono::high_resolution_clock::now();
        auto copy_duration = std::chrono::duration_cast<std::chrono::milliseconds>(copy_end - copy_start);
        std::cout << "Time taken by Memcopy: " << copy_duration.count() << " milliseconds" << std::endl;
        //////////

        hipFree(d_r_cutoff2);
        hipFree(d_xyz_1d);
        hipFree(d_neighborListArray);
        delete[] xyz_1d;
        delete[] temp_neighborListArray;
        // out();
    }
    // output the neighbor list array to a file
    void CudaCellList::out(){
        std::ofstream outfile;
        outfile.open("neb_list.txt");
        int neighborListArrayj = 0;
        for (int i = 0; i < _natoms; ++i) {
        // for (int i = 0; i < 10; ++i) {
            outfile << (i+1);
            for(int j = 0; j < 100; ++j) {
                neighborListArrayj = _neighborListArray[i * 100 + j];
                // outfile << "  " << (neighborListArrayj+1);
                if (neighborListArrayj >= 0 && neighborListArrayj != i) outfile << "\t" << (neighborListArrayj+1);
            }
            outfile << std::endl;
        }
        outfile.close();
    }

    // return the neighbor list array
    std::vector<std::vector<size_t>> CudaCellList::get_listArray() {
        std::vector<std::vector<size_t>> neighborListArray;
        for (int i = 0; i < _natoms; ++i) {
            std::vector<size_t> neighborListArrayi;
            for(int j = 0; j < 100; ++j) {
                int neighborListArrayj = _neighborListArray[i * 100 + j];
                if (neighborListArrayj >= 0 && neighborListArrayj != i) neighborListArrayi.push_back(neighborListArrayj);
            }
            neighborListArray.push_back(neighborListArrayi);
        }
        return neighborListArray;
    }

    // return the total time cost of the neighbor list array
    size_t CudaCellList::gettime() {
        return time_cost;
    }

    CudaCellList::~CudaCellList()
    {
        hipFree(d_box_len);
        hipFree(d_cell_len);
        hipFree(d_ncells);
        hipFree(d_natoms);
        hipFree(d_nebcell_list);
        hipFree(d_head);
        hipFree(d_lscl);
        hipFree(d_atom_cellindex);
        hipFree(d_cell_atoms_count);
        delete[] _cell_atoms_count;
        delete[] _neighborListArray;
    }


    base_NBL* NeighborList::createNeighborList(std::string type, Box *box, float r_cutoff, float skin)
    {
        Vec3<float> vec3_box_length = box->get_lengths();
        std::vector<float> box_length = {vec3_box_length[0], vec3_box_length[1], vec3_box_length[2]};
        if (type == "celllist") {
            //return new CudaCellList(box_length, r_cutoff, skin);
            return new CudaCellList(box, r_cutoff, skin);
        }
        else {
            std::cout << "type error" << std::endl;
            return NULL;
        }
    }

    NeighborList::~NeighborList()
    {
    }

}//namespace dpnblist

